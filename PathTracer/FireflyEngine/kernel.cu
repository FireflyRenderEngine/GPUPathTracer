#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "utilities.h"

#include <cmath>

surface<void, cudaSurfaceType2D> surf;

__device__ bool intersectPlane(const Geometry& plane, const Ray& ray, Intersect& intersect)
{
	// CLARIFICATION: all the rays need to be in object space; convert the ray to world space elsewhere
	float denom = glm::dot(plane.m_normal, ray.m_direction);
	if (glm::abs(denom) > 1e-7)
	{
		glm::vec3 p0l0 = -ray.m_origin;
		float t = glm::dot(p0l0, plane.m_normal) / denom;
		glm::vec3 P = ray.m_origin + t * ray.m_direction;
		// check bounds of the plane centered at 0,0,0 in object space
		if (!(P.x >= -0.5f && P.x <= 0.5f && P.y >= -0.5f && P.y <= 0.5f))
		{
			return false;
		}
	
		if (t > 0.0f) {
			intersect.m_t = t;
			intersect.m_intersectionPoint = P;
			intersect.m_normal = plane.m_normal;
			return true;
		}
		return false;
	}
	return false;
}

// fast Triangle intersection : https://cadxfem.org/inf/Fast%20MinimumStorage%20RayTriangle%20Intersection.pdf
__device__ bool intersectTriangle(const Triangle& triangle, const Ray& ray, Intersect& intersect)
{
	// CLARIFICATION: all the rays need to be in object space; convert the ray to world space elsewhere
	const float EPSILON = 0.000001;
	glm::vec3 vertex0 = triangle.m_v0;
	glm::vec3 vertex1 = triangle.m_v1;
	glm::vec3 vertex2 = triangle.m_v2;
	glm::vec3 edge1, edge2, pvec, tvec, qvec;
	float det, invDet, u, v;
	edge1 = vertex1 - vertex0;
	edge2 = vertex2 - vertex0;

	pvec = glm::cross(ray.m_direction, edge2);
	det = glm::dot(edge1, pvec);

	// BACKFACE CULLING
	if (det < EPSILON) {
		return false;    // This ray is parallel to this triangle.
	}

	tvec = ray.m_origin - vertex0;
	u = glm::dot(tvec, pvec);

	if (u < 0.0f || u > det) {
		return false;
	}

	qvec = glm::cross(tvec, edge1);

	v = glm::dot(ray.m_direction, qvec);
	if (v < 0.0f || u + v > det) {
		return false;
	}

	float t = glm::dot(edge2, qvec);

	invDet = 1.0 / det;

	t *= invDet;
	u *= invDet;
	v *= invDet;

	if (t > EPSILON) // ray intersection
	{
		glm::vec3 intersectPoint = ray.m_origin + ray.m_direction * t;
		intersect.m_intersectionPoint = intersectPoint;
		intersect.m_t = t;

		// Calculate the normal using barycentric coordinates
		float denom = (vertex1.y - vertex2.y) * (vertex0.x - vertex2.x) + (vertex2.x - vertex1.x) * (vertex0.y - vertex2.y);
		float wv1 = ((vertex1.y - vertex2.y) * (intersectPoint.x - vertex2.x) + (vertex2.x - vertex1.x) * (intersectPoint.y - vertex2.y)) / denom;
		float wv2 = ((vertex2.y - vertex0.y) * (intersectPoint.x - vertex2.x) + (vertex0.x - vertex2.x) * (intersectPoint.y - vertex2.y)) / denom;
		float wv3 = 1 - wv1 - wv2;
		intersect.m_normal = glm::normalize((wv1 * triangle.m_n0) + (wv2 * triangle.m_n1) + (wv3 * triangle.m_n2));
		return true;
	}
	else // This means that there is a line intersection but not a ray intersection.
	{
		return false;
	}
}

__device__ bool setIntersection(float& tMax, Intersect& intersectOut, const Intersect& objectSpaceIntersect, glm::mat4 invTransModelMatrix, glm::mat4 modelMatrix,const Ray& ray)
{
	// convert point of intersection into world space
	glm::vec3 worldPOI = modelMatrix * glm::vec4(objectSpaceIntersect.m_intersectionPoint, 1.0f);
	float distanceOfPOI = glm::distance(worldPOI, ray.m_origin);
	if (distanceOfPOI < tMax)
	{
		// right now we are storing the object space normal. Later on we calculate the world space normal.
		intersectOut.m_normal = objectSpaceIntersect.m_normal;
		// This is the world space point of intersection
		intersectOut.m_intersectionPoint = worldPOI;
		intersectOut.m_t = distanceOfPOI;
		intersectOut.m_hit = true;
		tMax = distanceOfPOI;
		return true;
	}
	return false;
}

__device__ Intersect& intersectRays(const Ray& ray, Geometry* geometries, unsigned int raytracableObjects)
{
	// This is the global intersect that stores the intersect info in world space
	Intersect intersectOut;
	float tMax = INFINITY;
	// loop through all geometries, find the smallest "t" value for a single ray
	for (int i = 0; i < raytracableObjects; ++i)
	{
		Geometry& geometry = geometries[i];

		// Generate the ray in the object space of the geometry being intersected.
		Ray& objectSpaceRay = Ray(geometry.m_inverseModelMatrix * glm::vec4(ray.m_origin, 1.f), glm::normalize(geometry.m_inverseModelMatrix * glm::vec4(ray.m_direction, 0.f)));

		// This intersect is re-created each iteration and stores the intersect info in object space of the geometry
		Intersect objectSpaceIntersect;

		if (geometry.m_geometryType == GeometryType::TRIANGLEMESH)
		{
			for (int j = 0; j < geometry.m_numberOfTriangles; ++j)
			{

				if (intersectTriangle(geometry.m_triangles[j], objectSpaceRay, objectSpaceIntersect))
				{
					if (setIntersection(tMax, intersectOut, objectSpaceIntersect, geometry.m_invTransModelMatrix, geometry.m_modelMatrix, ray)) 
					{
						intersectOut.geometryIndex = i;
						intersectOut.triangleIndex = j;
					}
				}
			}
		}
		else if (geometry.m_geometryType == GeometryType::PLANE)
		{
			if (intersectPlane(geometry, objectSpaceRay, objectSpaceIntersect))
			{
				if (setIntersection(tMax, intersectOut, objectSpaceIntersect, geometry.m_invTransModelMatrix, geometry.m_modelMatrix, ray)) 
				{
					// we store the geometry index so that we can access its BXDF later on
					intersectOut.geometryIndex = i;
				}
			}
		}
		else if (geometry.m_geometryType == GeometryType::SPHERE)
		{
			printf("Sphere Geometry implemented yet!");
		}
		else
		{
			printf("No such Geometry implemented yet!");
		}
	}

	// do matrix multiplication only once for calculating world space normal.
	// Instead of calculating a world space normal everytime a new tMax is found, do it only once at the end 
	if (intersectOut.m_hit)
	{
		intersectOut.m_normal = glm::normalize(glm::vec3(geometries[intersectOut.geometryIndex].m_invTransModelMatrix * glm::vec4(intersectOut.m_normal, 0.f)));
	}
	return intersectOut;
}

/**
 * @brief A helper function to call sampleBsdf of the intersected geometry.
	Same input/output parameters as BXDF::sampleBsdf. See utilities.h
*/
__device__ glm::vec3 getBXDF(const Ray& outgoingRay, const Intersect& intersect, glm::vec3& incomingRayDirection, Geometry* geometries, float& pdf, int depth, bool& isSpecular)
{
	return (geometries[intersect.geometryIndex].m_bxdf->sampleBsdf((-outgoingRay.m_direction), incomingRayDirection, intersect, pdf, depth, isSpecular));
}

/**
 * @brief generates a ray that starts from the location of the sensor/camera pointing forward
 * @param camera (INPUT): contains all the forward, up, near/far clip to construct the ray direction
 * @param x (INPUT): horizontal thread index for a certain pixel
 * @param y (INPUT): vertical thread index for a certain pixel
 * @param iterations (INPUT): the current iteration. Used to generate a unique seed for the random number generator
 * @return: a Ray which contains the origin and the newly generated direction
*/
__device__ Ray& generateRay(Camera camera, int x, int y, int iterations)
{
	Ray ray;

	// TODO: add depth of field
	ray.m_origin = camera.m_position;

	hiprandState state1;

	hiprand_init((unsigned long long)clock() + x, x, 0, &state1);
	float jx = hiprand_uniform(&state1);
	float jy = hiprand_uniform(&state1);

	// Stratified sample
	float Px = ((x + jx) / camera.m_screenWidth) * 2.f - 1.f;
	float Py = 1.f - ((y + jy) / camera.m_screenHeight) * 2.f;

	glm::vec3 wLookAtPoint = camera.m_invViewProj * (glm::vec4(Px, Py, 1.f, 1.f) * camera.m_farClip);

	ray.m_direction = glm::normalize(wLookAtPoint - ray.m_origin);
	return ray;
}

/**
 * @brief Each frame calls this kernel to trace rays into the scene. 
 *		  Each calls this kernel width*height times to fill the renderbuffer.
 *		  This kernel calculates the radiance generated by "totalSamplesPerPixel" number of rays at a pixel x,y.
 *		  This radiance calculated is filled into the CUDA surface object which is bound to the renderbuffer before kernel invocation.
 * @param geometries (INPUT): a simple array of geometries present in the scene.
 * @param lights (INPUT): a simple array of the indices at which lights are present in the geometries array.
 * @param camera (INPUT): the camera object that contains the location, forward, up, near/far clip etc to generate the starting ray to be traced
 * @param numberOfGeometries (INPUT): the total geometries present in the scene and the array "geometries". 
 *									  Since CUDA doesn't take in std::vector, we need to manually send the size of geometries
 * @param numberOfLights (INPUT): the total lights present in the scene and the array "lights". 
 *									  Since CUDA doesn't take in std::vector, we need to manually send the size of lights
 * @param iteration (INPUT): the current iteration. Used to average out the renderbuffer radiance (RGB) value.
 * @param maxDepth (INPUT): the maximum depth a ray is allowed to reach before terminating. 
 *							In complex scenes with lot of non-diffuse bsdfs, this needs to be high enough. But beware, the higher the maxDepth, the longer a path takes 
 * @param totalSamplesPerPixel (INPUT): to do anti aliasing. Shoots totalSamplesPerPixel for each kernel call to calculate radiance at pixel location.
 * @param d_pixelColor (INOUT): a device buffer to store radiance over iterations
 * @return : Fills the surface pointer that is bound to a certain (either FRONT or BACK) renderbuffer. This will be displayed by the framebuffer eventually
*/

__global__ void launchPathTrace(
	Geometry* geometries, 
	unsigned int* lights,
	Camera camera, 
	int numberOfGeometries, 
	int numberOfLights,
	int iteration,
	int maxDepth,
	int totalSamplesPerPixel,
	glm::vec3* d_pixelColor)
{
#ifdef PIXEL_DEBUG
	int x = 527;
	int y = 392;

#else
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
#endif
	int pixelSize = camera.m_screenHeight * camera.m_screenWidth;
	int pixelIndex = y * camera.m_screenWidth + x;

	if (pixelIndex >= pixelSize)
	{
		return;
	}
	// Do Light transport here
	// Loop over total number of samples to be shot per pixel (gives us anti aliasing)
	//   A. Loop until we hit max depth or russian roulette termination
	//		1. Check if we hit a light
	//		  1.a if we hit light, then terminate
	//		2. Check what material we hit
	//		  2.a get bsdf and pdf
	//		  2.b get next ray (incoming)
	//		  2.c calculate thruput and calculate russian roulette
	//		  2.d Go bath to A

	// This is where we will store the final radiance that will be converted to RGB
	// to be stored and displayed by the render buffer
	glm::vec3 finalPixelColor(0.f);

	// when we begin tracing rays, we need to clear & reset the render buffer (done outside this kernel)
	// and clear and reset the device buffer we use for accumulation.
	// This happens every time iteration is 1.
	if (iteration == 1)
	{
		d_pixelColor[pixelIndex] = glm::vec3(0.f);
	}

	finalPixelColor.x = d_pixelColor[pixelIndex].x;
	finalPixelColor.y = d_pixelColor[pixelIndex].y;
	finalPixelColor.z = d_pixelColor[pixelIndex].z;

	int samplesPerPixel = 1;
	glm::vec3 pixelColorPerPixel(0.f);
	
	while(samplesPerPixel <= totalSamplesPerPixel)
	{
		Ray& outgoingRay = generateRay(camera, x, y, iteration + samplesPerPixel);
		glm::vec3 pixelColorPerSample(0.f);
		int depth = 0;
		glm::vec3 thruput(1.f);

		bool lastSpecular = false;

		do
		{
			Intersect intersect = intersectRays(outgoingRay, geometries, numberOfGeometries);
			if (!intersect.m_hit)
			{
				break;
			}
			else
			{
				Ray incomingRay;
				incomingRay.m_origin = intersect.m_intersectionPoint;

				float pdf;
				// getBXDF returns 4 things: the bsdf, pdf of that bsdf sample, the new sampled direction, and if the bxdf is specular
				glm::vec3 bxdf = getBXDF(outgoingRay, intersect, incomingRay.m_direction, geometries, pdf, depth, lastSpecular);

				if (geometries[intersect.geometryIndex].m_bxdf->m_type == BXDFTyp::EMITTER)
				{
//#define NEE
#ifdef NEE
					if (depth > 0 && !lastSpecular)
					{
						break;
					}
					// add to thruput and exit since we hit an emitter
					pixelColorPerSample += thruput * bxdf;
					break;
#else
					pixelColorPerSample += thruput * bxdf;
					break;
#endif
				}

				if (pdf > RAY_EPSILON)
				{
					float dotProd = glm::abs(glm::dot(incomingRay.m_direction, intersect.m_normal));
					thruput *= dotProd * (bxdf / pdf);

#ifdef NEE
					if (geometries[intersect.geometryIndex].m_bxdf->m_type != BXDFTyp::MIRROR)
					{
						// NEE: we didn't hit a light, so we sample a point on a randomly selected light
						hiprandState state1;
						hiprandState state2;

						hiprand_init((unsigned long long)clock() + x, x, 0, &state1);
						unsigned int lightIdx = hiprand_uniform(&state1) * numberOfLights;


						hiprand_init((unsigned long long)clock() + y, y, 0, &state2);
						glm::vec2 sample(hiprand_uniform(&state1), hiprand_uniform(&state1));

						Intersect randomLightSample = geometries[lights[lightIdx]].sampleLight(sample);

						glm::vec3 shadowRayDirection = randomLightSample.m_intersectionPoint - intersect.m_intersectionPoint;

						float lengthSquared = glm::length(shadowRayDirection);
						lengthSquared *= lengthSquared;
						shadowRayDirection = glm::normalize(shadowRayDirection);
						float cosT = glm::dot(intersect.m_normal, shadowRayDirection);

						if (cosT > 0.f)
						{
							glm::vec3 originOffset = RAY_EPSILON * intersect.m_normal;
							Ray shadowRay(glm::dot(shadowRayDirection, originOffset) > 0 ? intersect.m_intersectionPoint + originOffset : intersect.m_intersectionPoint - originOffset, shadowRayDirection);
							Intersect lightIntersect = intersectRays(shadowRay, geometries, numberOfGeometries);

							if (lightIntersect.geometryIndex == lights[lightIdx] && geometries[lightIntersect.geometryIndex].m_bxdf->m_type == BXDFTyp::EMITTER)
							{
								float cosP = glm::dot(-shadowRayDirection, lightIntersect.m_normal);

								glm::vec3 lightBxdf = cosP > 0.f ? geometries[lightIntersect.geometryIndex].m_bxdf->m_emissiveColor * geometries[lightIntersect.geometryIndex].m_bxdf->m_intensity : glm::vec3(0.f);
								glm::vec3 directLighting = static_cast<float>(numberOfLights) * lightBxdf * cosT * cosP * geometries[lightIntersect.geometryIndex].m_surfaceArea / lengthSquared;
								pixelColorPerSample += directLighting * thruput;
							}
						}
					}
#endif
					
				}
				else
				{
					break;
				}
				// set the next ray for tracing
				glm::vec3 originOffset = RAY_EPSILON * intersect.m_normal;
				incomingRay.m_origin += glm::dot(incomingRay.m_direction, originOffset) > 0 ? originOffset : -originOffset;

				outgoingRay = incomingRay;
#define RR
#ifdef RR
				if (depth > 3)
				{
					hiprandState state;
					hiprand_init((unsigned long long)clock() + x, x, 0, &state);
					float q = glm::max(.05f, 1.f - thruput[1]);
					if (hiprand_uniform(&state) < q)
						break;
					thruput /= 1 - q;
				}
#endif
			}
			depth++;
		} while (depth < maxDepth);

		pixelColorPerPixel += pixelColorPerSample;
		
		samplesPerPixel++;
	}
	
	pixelColorPerPixel /= (float)(totalSamplesPerPixel);

	finalPixelColor += pixelColorPerPixel;
	
	d_pixelColor[pixelIndex] = finalPixelColor;	
	finalPixelColor /= iteration;

	// clamp the final rgb color [0, 1]
	finalPixelColor = glm::clamp(finalPixelColor, glm::vec3(0.f), glm::vec3(1.f));

	// write the color value to the pixel location x,y
	surf2Dwrite(make_uchar4(finalPixelColor[0] * 255, finalPixelColor[1] * 255, finalPixelColor[2] * 255, 255),
		surf,
		x * sizeof(uchar4),
		y,
		hipBoundaryModeZero);
}

hipError_t pxl_kernel_launcher(hipArray_const_t array,
	const int         width,
	const int         height,
	hipEvent_t       event,
	hipStream_t      stream,
	Geometry* geom,
	unsigned int* lights,
	Camera camera,
	int numGeom,
	int numLights,
	int iteration,
	int maxDepth,
	int samplesPerPixel,
	glm::vec3* d_pixelColor)
{
	hipError_t cuda_err;

	cuda_err = cudaBindSurfaceToArray(surf, array);

	if (cuda_err)
	{
		return cuda_err;
	}

	dim3 blockSize(16, 16, 1);
	dim3 gridSize;
	gridSize.x = ((width + blockSize.x - 1) / blockSize.x);
	gridSize.y = ((height + blockSize.y -1) / blockSize.y);
	
#ifdef PIXEL_DEBUG
	launchPathTrace << <1, 1, 0, stream >> > (geom, lights, camera, numGeom, numLights, iteration, maxDepth, samplesPerPixel, d_pixelColor);
#else
	launchPathTrace << <gridSize, blockSize, 0, stream >> > (geom, lights, camera, numGeom, numLights, iteration, maxDepth, samplesPerPixel, d_pixelColor);
#endif
	return hipSuccess;
}

int main()
{
	PathTracerState state;

	std::vector<Triangle> trianglesInMesh;
	LoadMesh(R"(..\..\sceneResources\sphere.obj)", trianglesInMesh);
	Geometry* triangleMeshGeometry = new Geometry("sphere",GeometryType::TRIANGLEMESH, glm::vec3(0.f, -0.5f, 0.f), glm::vec3(0.0f, 180.0f, 0.0f), glm::vec3(1.5f), trianglesInMesh);
	Geometry* topPlaneLightGeometry = new Geometry("ceiling light", GeometryType::PLANE, glm::vec3(0.f, 7.499f, 0.f), glm::vec3(90.f, 0.f, 0.f), glm::vec3(5.f));
	Geometry* leftPlaneLightGeometry = new Geometry("left light", GeometryType::PLANE, glm::vec3(-5.f, 0.f, 0.f), glm::vec3(0.f, 90.f, 0.f), glm::vec3(5.f));
	Geometry* bottomPlaneWhiteGeometry = new Geometry("floor", GeometryType::PLANE, glm::vec3(0.f, -7.5f, 0.f), glm::vec3(-90.f, 0.f, 0.f), glm::vec3(15.f));
	Geometry* topPlaneWhiteGeometry = new Geometry("ceiling", GeometryType::PLANE, glm::vec3(0.f, 7.5f, 0.f), glm::vec3(90.f, 0.f, 0.f), glm::vec3(15.f));
	Geometry* backPlaneWhiteGeometry = new Geometry("back plane", GeometryType::PLANE, glm::vec3(0.f, 0.f, -7.5f), glm::vec3(0.f), glm::vec3(15.f));
	Geometry* leftPlaneRedGeometry = new Geometry("red wall", GeometryType::PLANE, glm::vec3(-7.5f, 0.f, 0.f), glm::vec3(0.f, 90.f, 0.f), glm::vec3(15.f));
	Geometry* rightPlaneGreenGeometry = new Geometry("green wall", GeometryType::PLANE, glm::vec3(7.5f, 0.f, 0.f), glm::vec3(0.f, -90.f, 0.f), glm::vec3(15.f));


	BXDF* diffusebxdfREDMesh = new BXDF();
	diffusebxdfREDMesh->m_type = BXDFTyp::DIFFUSE;
	diffusebxdfREDMesh->m_albedo = { 1.f, 0.f, 0.f };

	BXDF* diffusebxdfGREENMesh = new BXDF();
	diffusebxdfGREENMesh->m_type = BXDFTyp::DIFFUSE;
	diffusebxdfGREENMesh->m_albedo = { 0.f, 1.f, 0.f };

	BXDF* diffusebxdfBLUEMesh = new BXDF();
	diffusebxdfBLUEMesh->m_type = BXDFTyp::DIFFUSE;
	diffusebxdfBLUEMesh->m_albedo = { 0.f, 0.f, 1.f };

	BXDF* diffusebxdfPURPLEMesh = new BXDF();
	diffusebxdfPURPLEMesh->m_type = BXDFTyp::DIFFUSE;
	diffusebxdfPURPLEMesh->m_albedo = { 1.f, 0.f, 1.f };

	BXDF* diffusebxdfWHITEMesh = new BXDF();
	diffusebxdfWHITEMesh->m_type = BXDFTyp::DIFFUSE;
	diffusebxdfWHITEMesh->m_albedo = { 1.f, 1.f, 1.f };

	BXDF* lightbxdfPlane = new BXDF();
	lightbxdfPlane->m_type = BXDFTyp::EMITTER;
	lightbxdfPlane->m_intensity = 1.0f;
	lightbxdfPlane->m_emissiveColor = { 1.f, 1.f, 1.f };

	BXDF* specularbxdfWHITEMesh = new BXDF();
	specularbxdfWHITEMesh->m_type = BXDFTyp::MIRROR;
	specularbxdfWHITEMesh->m_specularColor = { 1.f, 1.f, 1.f };

	triangleMeshGeometry->m_bxdf = specularbxdfWHITEMesh;
	bottomPlaneWhiteGeometry->m_bxdf = diffusebxdfWHITEMesh;
	backPlaneWhiteGeometry->m_bxdf = diffusebxdfWHITEMesh;
	topPlaneWhiteGeometry->m_bxdf = diffusebxdfWHITEMesh;
	leftPlaneRedGeometry->m_bxdf = diffusebxdfREDMesh;
	rightPlaneGreenGeometry->m_bxdf = diffusebxdfGREENMesh;
	topPlaneLightGeometry->m_bxdf = lightbxdfPlane;
	leftPlaneLightGeometry->m_bxdf = lightbxdfPlane;
	
	std::vector<Geometry> geometries;
	geometries.push_back(*triangleMeshGeometry);
	geometries.push_back(*topPlaneLightGeometry);
	//geometries.push_back(*leftPlaneLightGeometry);
	geometries.push_back(*bottomPlaneWhiteGeometry);
	geometries.push_back(*backPlaneWhiteGeometry);
	geometries.push_back(*topPlaneWhiteGeometry);
	geometries.push_back(*rightPlaneGreenGeometry);
	geometries.push_back(*leftPlaneRedGeometry);

	std::vector<unsigned int> lights;
	for (unsigned int i = 0; i < geometries.size(); ++i)
	{
		if (geometries[i].m_bxdf->m_type == BXDFTyp::EMITTER)
		{
			lights.push_back(i);
		}
	}

	// First we will copy the base geometry object to device memory
	unsigned int* d_lights = nullptr;
	hipMalloc((void**)&(d_lights), sizeof(unsigned int) * lights.size());
	cudaCheckErrors("hipMalloc lights fail");
	hipMemcpy(d_lights, lights.data(), sizeof(unsigned int) * lights.size(), hipMemcpyHostToDevice);
	cudaCheckErrors("hipMemcpy lights fail");

	// TODO: Load scene from file
	int windowWidth  = 800;
	int windowHeight = 800;
	int cameraResolution = windowWidth * windowHeight;

	// First we will copy the base geometry object to device memory
	state.d_geometry = nullptr;
	hipMalloc((void**)&(state.d_geometry), sizeof(Geometry) * geometries.size());
	cudaCheckErrors("hipMalloc geometry fail");
	hipMemcpy(state.d_geometry, geometries.data(), sizeof(Geometry) * geometries.size(), hipMemcpyHostToDevice);
	cudaCheckErrors("hipMemcpy geometry fail");
	state.d_raytracableObjects = geometries.size();

	// Now we will save the internal triangle data to device memory
	for (int i = 0; i < geometries.size(); ++i)
	{
		BXDF* hostBXDFData;
		hipMallocManaged((void**)&hostBXDFData, sizeof(BXDF));
		cudaCheckErrors("hipMalloc host bxdf data fail");
		hipMemcpy(hostBXDFData, geometries[i].m_bxdf, sizeof(BXDF), hipMemcpyHostToDevice);
		cudaCheckErrors("hipMemcpy host bxdf data fail");
		hipMemcpy(&(state.d_geometry[i].m_bxdf), &hostBXDFData, sizeof(BXDF*), hipMemcpyHostToDevice);
		cudaCheckErrors("hipMemcpy device bxdf data fail");

#ifdef PIXEL_DEBUG
		const char* hostNames;
		hipMallocManaged((void**)&hostNames, sizeof(const char*));
		cudaCheckErrors("hipMalloc host name data fail");
		hipMemcpy(const_cast<char*>(hostNames), geometries[i].m_name, sizeof(const char*), hipMemcpyHostToDevice);
		cudaCheckErrors("hipMemcpy host name data fail");
		hipMemcpy(&(state.d_geometry[i].m_name), &hostNames, sizeof(const char*), hipMemcpyHostToDevice);
		cudaCheckErrors("hipMemcpy device name data fail");
#endif

		if (geometries[i].m_geometryType == GeometryType::TRIANGLEMESH)
		{
			// TODO: Figure out a better way to allocate and deallocate this hostTriangleData
			Triangle* hostTriangleData;
			hipMallocManaged((void**)&hostTriangleData, sizeof(Triangle) * geometries[i].m_numberOfTriangles);
			cudaCheckErrors("hipMalloc host triangle data fail");
			hipMemcpy(hostTriangleData, geometries[i].m_triangles, sizeof(Triangle) * geometries[i].m_numberOfTriangles, hipMemcpyHostToDevice);
			cudaCheckErrors("hipMemcpy host triangle data fail");
			hipMemcpy(&(state.d_geometry[i].m_triangles), &hostTriangleData, sizeof(Triangle*), hipMemcpyHostToDevice);
			cudaCheckErrors("hipMemcpy device triangle data fail");
		}
	}

	Camera camera;
	camera.m_position = glm::vec3(0.f, 0.f, 16.5f);
	camera.m_forward = glm::vec3(0.f, 0.f, -1.f);
	camera.m_worldUp = glm::vec3(0.f, 1.f, 0.f);
	camera.m_fov = 70.f;
	camera.m_screenHeight = float(windowWidth);
	camera.m_screenWidth = float(windowHeight);
	camera.m_nearClip = 0.001f;
	camera.m_farClip = 10000.f;
	camera.m_pitch = 0.f;
	camera.m_yaw = -90.f;
	camera.UpdateBasisAxis();

	camera.m_invViewProj = camera.GetInverseViewMatrix() * camera.GetInverseProjectionMatrix();

	GLFWViewer* viewer = new GLFWViewer(windowWidth, windowHeight);

	glm::vec3* d_pixelColor = nullptr;
	hipMalloc((void**)&(d_pixelColor), sizeof(glm::vec3) * windowWidth * windowHeight);
	cudaCheckErrors("hipMalloc d_pixelColor fail");

	state.d_camera = nullptr;
	hipMalloc((void**)&(state.d_camera), sizeof(Camera));
	cudaCheckErrors("hipMalloc camera fail");

	int iteration = 1;

	int maxDepth = 6;
	int samplesPerPixel = 4;

	GpuTimer timer;
	float time = 0.f;

	while (!glfwWindowShouldClose(viewer->m_window))
	{
		processInput(viewer->m_window, camera, viewer, iteration, time);
		camera.m_invViewProj = camera.GetInverseViewMatrix() * camera.GetInverseProjectionMatrix();

		//
		// EXECUTE CUDA KERNEL ON RENDER BUFFER
		//

		hipGraphicsMapResources(1, &viewer->interop->cgr[viewer->interop->index], viewer->stream);
		{
			timer.Start();
			viewer->cuda_err = pxl_kernel_launcher(viewer->interop->ca[viewer->interop->index] ,
				windowWidth,
				windowHeight,
				viewer->event,
				viewer->stream,
				state.d_geometry, 
				d_lights,
				camera, 
				geometries.size(),
				lights.size(),
				iteration, 
				maxDepth,
				samplesPerPixel,
				d_pixelColor);
			timer.Stop();
		}
		hipGraphicsUnmapResources(1, &viewer->interop->cgr[viewer->interop->index], viewer->stream);

		char title[256];
		time = timer.Elapsed();
		sprintf(title, "Firefly | FPS %f | iteration: %d | kernel took: %.2fs | samples per pixel: %d | max depth: %d", 1.0f/time, iteration, time/iteration, samplesPerPixel, maxDepth);
		glfwSetWindowTitle(viewer->m_window, title);
		
		if (iteration == 16)
		{
			//saveToPPM(viewer);
		}

		//
		// BLIT & SWAP FBO
		// 
		glBlitNamedFramebuffer(viewer->interop->fb[viewer->interop->index], 0,
			0, 0, viewer->interop->width, viewer->interop->height,
			0, viewer->interop->height, viewer->interop->width, 0,
			GL_COLOR_BUFFER_BIT,
			GL_NEAREST);

		viewer->interop->index = (viewer->interop->index + 1) % viewer->interop->count;
		iteration++;

		glfwSwapBuffers(viewer->m_window);
		glfwPollEvents();
	}

	glfwDestroyWindow(viewer->m_window);
	glfwTerminate();

	hipFree(state.d_geometry);
	delete viewer;
	hipFree(d_pixelColor);
	return 0;
}
