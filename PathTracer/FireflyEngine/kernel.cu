#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "utilities.h"

__global__ void generateRays(Ray* rays, Camera* camera)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int pixelSize = camera->m_screenHeight * camera->m_screenWidth;
	int pixelIndex = y * camera->m_screenWidth + x;

	if (pixelIndex >= pixelSize)
	{
		return;
	}
	Ray& ray = rays[pixelIndex];
	ray.m_origin = camera->m_position;

	float Px = (x / camera->m_screenWidth) * 2.f - 1.f;
	float Py = 1.f - (y / camera->m_screenHeight) * 2.f;

	glm::vec3 wLookAtPoint = camera->GetInverseViewMatrix() * camera->GetInverseProjectionMatrix() * (glm::vec4(Px, Py, 1.f, 1.f) * camera->m_farClip);

	ray.m_direction = glm::normalize(wLookAtPoint - ray.m_origin);
}

__device__ bool intersectTriangle(const Triangle& triangle, const Ray& ray, Intersect& intersect)
{
	const float EPSILON = 0.0000001;
	glm::vec3 vertex0 = triangle.m_v0;
	glm::vec3 vertex1 = triangle.m_v1;
	glm::vec3 vertex2 = triangle.m_v2;
	glm::vec3 edge1, edge2, h, s, q;
	float a, f, u, v;
	edge1 = vertex1 - vertex0;
	edge2 = vertex2 - vertex0;
	h = glm::cross(ray.m_direction, edge2);
	a = glm::dot(edge1, h);
	if (a > -EPSILON && a < EPSILON)
	{
		return false;    // This ray is parallel to this triangle.
	}
	f = 1.0 / a;
	s = ray.m_origin - vertex0;
	u = f * glm::dot(s, h);
	if (u < 0.0 || u > 1.0)
	{
		//ray.m_t = 0.4f;
		return false;
	}
	q = glm::cross(s, edge1);
	v = f * glm::dot(ray.m_direction, q);
	if (v < 0.0 || u + v > 1.0)
		return false;
	// At this stage we can compute t to find out where the intersection point is on the line.
	float t = f * glm::dot(edge2, q);
	if (t > EPSILON) // ray intersection
	{
		intersect.m_intersectionPoint = ray.m_origin + ray.m_direction * t;
		intersect.m_t = t;
		return true;
	}
	else // This means that there is a line intersection but not a ray intersection.
	{
		return false;
	}
}

__global__ void intersectRays(Camera* camera, Ray* rays, Triangle* triangles, glm::vec3* pixels)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int pixelSize = camera->m_screenWidth * camera->m_screenHeight;
	int pixelIndex = y * camera->m_screenWidth + x;

	if (pixelIndex >= pixelSize)
	{
		return;
	}

	Intersect intersect;

	for (int i = 0; i < 10; ++i) 
	{
		if (intersectTriangle(triangles[i], rays[pixelIndex], intersect)) 
		{
			pixels[pixelIndex] = glm::vec3(255, 0.0f, 0.0f);
			continue;
		}
	}
}

int main()
{
	// Load a triangle mesh
	std::vector<Triangle*> trianglesInMesh;
	for (int i = 0 ; i < 10; ++i)
	{
		Triangle* triangle = new Triangle(
			glm::vec3(i, i, i),
			glm::vec3(i + 1.f, i + 1.f, 0.0f),
			glm::vec3(i + 2.f, 0.0f, 0.0f), 
			glm::vec2(0.0f), glm::vec2(0.0f), glm::vec2(0.0f),
			glm::vec3(0.0f), glm::vec3(0.0f), glm::vec3(0.0f));
		trianglesInMesh.push_back(triangle);
	}
	Mesh* triangleMesh = new Mesh(2, glm::vec3(0.0f), glm::vec3(0.0f), glm::vec3(1.0f), trianglesInMesh);

	int windowWidth = 800;
	int windowHeight = 800;
	int dataSize = windowWidth * windowHeight;

	Triangle* d_triangle = nullptr;
	hipMalloc((void**)&d_triangle, sizeof(Triangle) * triangleMesh->m_numberOfTriangles);

	Ray* d_rays = nullptr;
	hipMalloc((void**)&d_rays, dataSize * sizeof(Ray));

	glm::vec3* pixels = new glm::vec3[dataSize];
	// Initialize all the pixels with a base color of white
	for (int i = 0 ; i < dataSize; ++i) 
	{
		pixels[i] = glm::vec3(255.f, 255.f, 255.f);
	}
	glm::vec3* d_pixels = nullptr;
	hipMalloc((void**)&d_pixels, dataSize * sizeof(glm::vec3));
	hipMemcpy(d_pixels, pixels, dataSize * sizeof(glm::vec3), hipMemcpyHostToDevice);

	dim3 blockSize(16, 16, 1);
	dim3 gridSize;
	gridSize.x = (windowWidth / blockSize.x);// +1;
	gridSize.y = (windowHeight / blockSize.y);// +1;

	Camera* camera = new Camera();
	camera->m_position = glm::vec3(0.f, 5.f, 15.f);
	camera->m_forward = glm::vec3(0.f, 0.f, -1.f);
	camera->m_worldUp = glm::vec3(0.f, 1.f, 0.f);
	camera->m_fov = 70.f;
	camera->m_screenHeight = float(windowWidth);
	camera->m_screenWidth = float(windowHeight);
	camera->m_nearClip = 0.1f;
	camera->m_farClip = 1000.f;
	camera->m_pitch = 0.f;
	camera->m_yaw = -90.f;
	camera->UpdateBasisAxis();

	GLFWViewer* viewer = new GLFWViewer(windowWidth, windowHeight, pixels);
	viewer->Create();

	Camera* d_camera = nullptr;
	hipMalloc((void**)&d_camera, sizeof(Camera));

	hipMemcpy(d_triangle, triangleMesh->m_triangles, sizeof(Triangle) * triangleMesh->m_numberOfTriangles, hipMemcpyHostToDevice);

	while (!glfwWindowShouldClose(viewer->m_window))
	{
		processInput(viewer->m_window, camera, pixels);
		hipMemcpy(d_camera, camera, sizeof(Camera), hipMemcpyHostToDevice);
		generateRays << <gridSize, blockSize >> > (d_rays, d_camera);
		// Initialize all the pixels with a base color of white
		for (int i = 0; i < dataSize; ++i)
		{
			pixels[i] = glm::vec3(255.f, 255.f, 255.f);
		}
		hipMemcpy(d_pixels, pixels, dataSize * sizeof(glm::vec3), hipMemcpyHostToDevice);
		intersectRays << <gridSize, blockSize >> > (d_camera, d_rays, d_triangle, d_pixels);
		hipMemcpy(pixels, d_pixels, sizeof(glm::vec3) * dataSize, hipMemcpyDeviceToHost);

		glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);

		viewer->Draw();

		glfwSwapBuffers(viewer->m_window);
		glfwPollEvents();
	}
	
	//Ray* rays = new Ray[dataSize];
	//hipMemcpy(rays, d_rays, dataSize * sizeof(Ray), hipMemcpyDeviceToHost);

	delete pixels;
	return 0;
}
