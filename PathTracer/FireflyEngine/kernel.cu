#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "utilities.h"

#include <cmath>

surface<void, cudaSurfaceType2D> surf;

__device__ bool intersectPlane(const Geometry& plane, const Ray& ray, Intersect& intersect)
{
	// CLARIFICATION: all the rays need to be in object space; convert the ray to world space elsewhere
	float denom = glm::dot(plane.m_normal, ray.m_direction);
	if (glm::abs(denom) > 1e-7)
	{
		glm::vec3 p0l0 = -ray.m_origin;
		float t = glm::dot(p0l0, plane.m_normal) / denom;
		glm::vec3 P = ray.m_origin + t * ray.m_direction;
		// check bounds of the plane centered at 0,0,0 in object space
		if (!(P.x >= -0.5f && P.x <= 0.5f && P.y >= -0.5f && P.y <= 0.5f))
		{
			return false;
		}
	
		if (t > 0.0f) {
			intersect.m_t = t;
			intersect.m_intersectionPoint = P;
			intersect.m_normal = plane.m_normal;
			return true;
		}
		return false;
	}
	return false;
}

// fast Triangle intersection : https://cadxfem.org/inf/Fast%20MinimumStorage%20RayTriangle%20Intersection.pdf
__device__ bool intersectTriangle(const Triangle& triangle, const Ray& ray, Intersect& intersect)
{
	// CLARIFICATION: all the rays need to be in object space; convert the ray to world space elsewhere
	const float EPSILON = 0.000001;
	glm::vec3 vertex0 = triangle.m_v0;
	glm::vec3 vertex1 = triangle.m_v1;
	glm::vec3 vertex2 = triangle.m_v2;
	glm::vec3 edge1, edge2, pvec, tvec, qvec;
	float det, invDet, u, v;
	edge1 = vertex1 - vertex0;
	edge2 = vertex2 - vertex0;

	pvec = glm::cross(ray.m_direction, edge2);
	det = glm::dot(edge1, pvec);

	// BACKFACE CULLING
	if (det < EPSILON) {
		return false;    // This ray is parallel to this triangle.
	}

	tvec = ray.m_origin - vertex0;
	u = glm::dot(tvec, pvec);

	if (u < 0.0f || u > det) {
		return false;
	}

	qvec = glm::cross(tvec, edge1);

	v = glm::dot(ray.m_direction, qvec);
	if (v < 0.0f || u + v > det) {
		return false;
	}

	float t = glm::dot(edge2, qvec);

	invDet = 1.0 / det;

	t *= invDet;
	u *= invDet;
	v *= invDet;

	if (t > EPSILON) // ray intersection
	{
		glm::vec3 intersectPoint = ray.m_origin + ray.m_direction * t;
		intersect.m_intersectionPoint = intersectPoint;
		intersect.m_t = t;

		// Calculate the normal using barycentric coordinates
		float denom = (vertex1.y - vertex2.y) * (vertex0.x - vertex2.x) + (vertex2.x - vertex1.x) * (vertex0.y - vertex2.y);
		float wv1 = ((vertex1.y - vertex2.y) * (intersectPoint.x - vertex2.x) + (vertex2.x - vertex1.x) * (intersectPoint.y - vertex2.y)) / denom;
		float wv2 = ((vertex2.y - vertex0.y) * (intersectPoint.x - vertex2.x) + (vertex0.x - vertex2.x) * (intersectPoint.y - vertex2.y)) / denom;
		float wv3 = 1 - wv1 - wv2;
		intersect.m_normal = glm::normalize((wv1 * triangle.m_n0) + (wv2 * triangle.m_n1) + (wv3 * triangle.m_n2));
		return true;
	}
	else // This means that there is a line intersection but not a ray intersection.
	{
		return false;
	}
}

__device__ bool setIntersection(float& tMax, Intersect& intersectOut, const Intersect& objectSpaceIntersect, glm::mat4 invTransModelMatrix, glm::mat4 modelMatrix,const Ray& ray)
{
	// convert point of intersection into world space
	glm::vec3 worldPOI = modelMatrix * glm::vec4(objectSpaceIntersect.m_intersectionPoint, 1.0f);
	float distanceOfPOI = glm::distance(worldPOI, ray.m_origin);
	if (distanceOfPOI < tMax)
	{
		// right now we are storing the object space normal. Later on we calculate the world space normal.
		intersectOut.m_normal = objectSpaceIntersect.m_normal;
		// This is the world space point of intersection
		intersectOut.m_intersectionPoint = worldPOI;
		intersectOut.m_t = distanceOfPOI;
		intersectOut.m_hit = true;
		tMax = distanceOfPOI;
		return true;
	}
	return false;
}

__device__ Intersect& intersectRays(const Ray& ray, Geometry* geometries, unsigned int raytracableObjects)
{
	// This is the global intersect that stores the intersect info in world space
	Intersect intersectOut;
	float tMax = INFINITY;
	// loop through all geometries, find the smallest "t" value for a single ray
	for (int i = 0; i < raytracableObjects; ++i)
	{
		Geometry& geometry = geometries[i];

		// Generate the ray in the object space of the geometry being intersected.
		Ray& objectSpaceRay = Ray(geometry.m_inverseModelMatrix * glm::vec4(ray.m_origin, 1.f), glm::normalize(geometry.m_inverseModelMatrix * glm::vec4(ray.m_direction, 0.f)));

		// This intersect is re-created each iteration and stores the intersect info in object space of the geometry
		Intersect objectSpaceIntersect;

		if (geometry.m_geometryType == GeometryType::TRIANGLEMESH)
		{
			for (int j = 0; j < geometry.m_numberOfTriangles; ++j)
			{

				if (intersectTriangle(geometry.m_triangles[j], objectSpaceRay, objectSpaceIntersect))
				{
					if (setIntersection(tMax, intersectOut, objectSpaceIntersect, geometry.m_invTransModelMatrix, geometry.m_modelMatrix, ray)) 
					{
						intersectOut.geometryIndex = i;
						intersectOut.triangleIndex = j;
					}
				}
			}
		}
		else if (geometry.m_geometryType == GeometryType::PLANE)
		{
			if (intersectPlane(geometry, objectSpaceRay, objectSpaceIntersect))
			{
				if (setIntersection(tMax, intersectOut, objectSpaceIntersect, geometry.m_invTransModelMatrix, geometry.m_modelMatrix, ray)) 
				{
					// we store the geometry index so that we can access its BXDF later on
					intersectOut.geometryIndex = i;
				}
			}
		}
		else if (geometry.m_geometryType == GeometryType::SPHERE)
		{
			printf("Sphere Geometry implemented yet!");
		}
		else
		{
			printf("No such Geometry implemented yet!");
		}
	}

	// do matrix multiplication only once for calculating world space normal.
	// Instead of calculating a world space normal everytime a new tMax is found, do it only once at the end 
	if (intersectOut.m_hit)
	{
		intersectOut.m_normal = glm::normalize(glm::vec3(geometries[intersectOut.geometryIndex].m_invTransModelMatrix * glm::vec4(intersectOut.m_normal, 0.f)));
	}
	return intersectOut;
}

/**
 * @brief A helper function to call sampleBsdf of the intersected geometry.
	Same input/output parameters as BXDF::sampleBsdf. See utilities.h
*/
__device__ glm::vec3 getBXDF(const Ray& outgoingRay, const Intersect& intersect, glm::vec3& incomingRayDirection, Geometry* geometries, float& pdf, int depth)
{
	return (geometries[intersect.geometryIndex].m_bxdf->sampleBsdf((-outgoingRay.m_direction), incomingRayDirection, intersect, pdf, depth));
}

/**
 * @brief generates a ray that starts from the location of the sensor/camera pointing forward
 * @param camera (INPUT): contains all the forward, up, near/far clip to construct the ray direction
 * @param x (INPUT): horizontal thread index for a certain pixel
 * @param y (INPUT): vertical thread index for a certain pixel
 * @param iterations (INPUT): the current iteration. Used to generate a unique seed for the random number generator
 * @return: a Ray which contains the origin and the newly generated direction
*/
__device__ Ray& generateRay(Camera camera, int x, int y, int iterations)
{
	Ray ray;

	// TODO: add depth of field
	ray.m_origin = camera.m_position;

	hiprandState state1;
	hiprandState state2;

	hiprand_init((unsigned long long)clock() + x, x, 0, &state1);
	float jx = hiprand_uniform(&state1);
	float jy = hiprand_uniform(&state1);

	// Stratified sample
	float Px = ((x + jx) / camera.m_screenWidth) * 2.f - 1.f;
	float Py = 1.f - ((y + jy) / camera.m_screenHeight) * 2.f;

	glm::vec3 wLookAtPoint = camera.m_invViewProj * (glm::vec4(Px, Py, 1.f, 1.f) * camera.m_farClip);

	ray.m_direction = glm::normalize(wLookAtPoint - ray.m_origin);
	return ray;
}

/**
 * @brief Each frame calls this kernel to trace rays into the scene. 
 *		  Each calls this kernel width*height times to fill the renderbuffer.
 *		  This kernel calculates the radiance generated by "totalSamplesPerPixel" number of rays at a pixel x,y.
 *		  This radiance calculated is filled into the CUDA surface object which is bound to the renderbuffer before kernel invocation.
 * @param geometries (INPUT): a simple array of geometries present in the scene.
 * @param camera (INPUT): the camera object that contains the location, forward, up, near/far clip etc to generate the starting ray to be traced
 * @param numberOfGeometries (INPUT): the total geometries present in the scene and the array "geometries". 
 *									  Since CUDA doesn't take in std::vector, we need to manually send the size of geometries
 * @param iteration (INPUT): the current iteration. Used to average out the renderbuffer radiance (RGB) value.
 * @param maxDepth (INPUT): the maximum depth a ray is allowed to reach before terminating. 
 *							In complex scenes with lot of non-diffuse bsdfs, this needs to be high enough. But beware, the higher the maxDepth, the longer a path takes 
 * @param totalSamplesPerPixel (INPUT): to do anti aliasing. Shoots totalSamplesPerPixel for each kernel call to calculate radiance at pixel location.
 * @param d_pixelColor (INOUT): a device buffer to store radiance over iterations
 * @return : Fills the surface pointer that is bound to a certain (either FRONT or BACK) renderbuffer. This will be displayed by the framebuffer eventually
*/

__global__ void launchPathTrace(
	Geometry* geometries, 
	Camera camera, 
	int numberOfGeometries, 
	int iteration,
	int maxDepth,
	int totalSamplesPerPixel,
	glm::vec3* d_pixelColor)
{
	int x = /*200; */blockIdx.x* blockDim.x + threadIdx.x;
	int y = /*400; */blockIdx.y* blockDim.y + threadIdx.y;

	int pixelSize = camera.m_screenHeight * camera.m_screenWidth;
	int pixelIndex = y * camera.m_screenWidth + x;

	if (pixelIndex >= pixelSize)
	{
		return;
	}
	// Do Light transport here
	// Loop over total number of samples to be shot per pixel (gives us anti aliasing)
	//   A. Loop until we hit max depth or russian roulette termination
	//		1. Check if we hit a light
	//		  1.a if we hit light, then terminate
	//		2. Check what material we hit
	//		  2.a get bsdf and pdf
	//		  2.b get next ray (incoming)
	//		  2.c calculate thruput and calculate russian roulette
	//		  2.d Go bath to A

	// This is where we will store the final radiance that will be converted to RGB
	// to be stored and displayed by the render buffer
	glm::vec3 finalPixelColor(0.f);

	// when we begin tracing rays, we need to clear & reset the render buffer (done outside this kernel)
	// and clear and reset the device buffer we use for accumulation.
	// This happens every time iteration is 1.
	if (iteration == 1)
	{
		d_pixelColor[pixelIndex] = glm::vec3(0.f);
	}

	finalPixelColor.x = d_pixelColor[pixelIndex].x;
	finalPixelColor.y = d_pixelColor[pixelIndex].y;
	finalPixelColor.z = d_pixelColor[pixelIndex].z;

	int samplesPerPixel = 1;
	int pathsThatContributed = 0;
	glm::vec3 pixelColorPerPixel(0.f);
	
	while(samplesPerPixel <= totalSamplesPerPixel)
	{
		Ray& outgoingRay = generateRay(camera, x, y, iteration + samplesPerPixel);
		glm::vec3 pixelColorPerSample(0.f);
		int depth = 0;
		glm::vec3 thruput(1.f);

		do
		{
			Intersect intersect = intersectRays(outgoingRay, geometries, numberOfGeometries);
			if (!intersect.m_hit)
			{
				break;
			}
			else
			{
				Ray incomingRay;
				incomingRay.m_origin = intersect.m_intersectionPoint;

				float pdf;
				// getBXDF returns 3 things: the bsdf, pdf of that bsdf sample, and the new sampled direction
				glm::vec3 bxdf = getBXDF(outgoingRay, intersect, incomingRay.m_direction, geometries, pdf, depth);

				if (geometries[intersect.geometryIndex].m_bxdf->m_type == BXDFTyp::EMITTER)
				{
					// add to thruput and exit since we hit an emitter
					pixelColorPerSample += thruput * bxdf;
					pathsThatContributed++;
					break;
				}

				if (pdf > 0.0001f)
				{
					float dotProd = glm::abs(glm::dot(incomingRay.m_direction, intersect.m_normal));
					thruput *= dotProd * (bxdf / pdf);
				}
				else
				{
					break;
				}
				// set the next ray for tracing
				glm::vec3 originOffset = 0.005f * intersect.m_normal;
				incomingRay.m_origin += glm::dot(incomingRay.m_direction, originOffset) > 0 ? originOffset : -originOffset;

				outgoingRay = incomingRay;
			}
			depth++;
		} while (depth < maxDepth);

		pixelColorPerPixel += pixelColorPerSample;
		
		samplesPerPixel++;
	}
	
	if(pathsThatContributed > 0)
	{
		pixelColorPerPixel /= (float)(totalSamplesPerPixel);
	}

	finalPixelColor += pixelColorPerPixel;
	
	d_pixelColor[pixelIndex] = finalPixelColor;	
	finalPixelColor /= iteration;

	// clamp the final rgb color [0, 1]
	finalPixelColor = glm::vec3(glm::clamp(finalPixelColor.x, 0.f, 1.f), glm::clamp(finalPixelColor.y, 0.f, 1.f), glm::clamp(finalPixelColor.z, 0.f, 1.f));

	// write the color value to the pixel location x,y
	surf2Dwrite(make_uchar4(finalPixelColor[0] * 255, finalPixelColor[1] * 255, finalPixelColor[2] * 255, 255),
		surf,
		x * sizeof(uchar4),
		y,
		hipBoundaryModeZero);
}

hipError_t pxl_kernel_launcher(hipArray_const_t array,
	const int         width,
	const int         height,
	hipEvent_t       event,
	hipStream_t      stream,
	Geometry* geom,
	Camera camera,
	int numGeom,
	int iteration,
	int maxDepth,
	int samplesPerPixel,
	glm::vec3* d_pixelColor)
{
	hipError_t cuda_err;

	cuda_err = cudaBindSurfaceToArray(surf, array);

	if (cuda_err)
	{
		return cuda_err;
	}

	dim3 blockSize(16, 16, 1);
	dim3 gridSize;
	gridSize.x = ((width + blockSize.x - 1) / blockSize.x);
	gridSize.y = ((height + blockSize.y -1) / blockSize.y);
	
	launchPathTrace << </*1,1*/gridSize, blockSize, 0, stream >> > (geom, camera, numGeom, iteration, maxDepth, samplesPerPixel, d_pixelColor);

	return hipSuccess;
}

__global__ void CreateWorldAABB(Geometry* geometries, size_t numGeometries, int numOfPrimitives, AABB* AABBArray) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (index > numOfPrimitives) {
		return;
	}

	int primitiveCount = 0;
	for (int i = 0; i < numGeometries; ++i)
	{
		primitiveCount += (geometries[i].m_geometryType == GeometryType::TRIANGLEMESH) ? geometries[i].m_numberOfTriangles : 1;
		if (index < primitiveCount) {
			if (geometries[i].m_geometryType != GeometryType::TRIANGLEMESH) {
				 BuildAABB(AABBArray[index], geometries[i], i, -1);
			}
			else {
				 BuildAABB(AABBArray[index], geometries[i], i, index - i);
			}
		}
	}

	//int primitiveIndex = 0;
	//for (int i = 0; i < numGeometries; ++i)
	//{
	//	if (index < numGeometries)
	//	{
	//		if (geometries[i].m_geometryType != GeometryType::TRIANGLEMESH)
	//		{
	//			if (index == i)
	//			{
	//				// need to break here
	//				BuildAABB(geometries[i], -1);
	//			}
	//			continue;
	//		}
	//		else
	//		{
	//			// need to break here
	//			BuildAABB(geometries[i], index);
	//			break;
	//		}
	//	}
	//	else
	//	{
	//		if (geometries[i].m_geometryType != GeometryType::TRIANGLEMESH)
	//		{
	//			if (primitiveIndex > 0)
	//			{
	//				if (index == primitiveIndex)
	//				{
	//					// found; need to break here
	//					BuildAABB(geometries[i], -1);
	//					break;
	//				}
	//				else
	//				{
	//					primitiveIndex++;
	//				}
	//			}
	//			continue;
	//		}
	//		else
	//		{
	//			if ((geometries[i].m_numTriangles + i + primitiveIndex) <= index)
	//			{
	//				primitiveIndex = geometries[i].m_numTriangles + i;
	//				continue;
	//			}
	//			else
	//			{
	//				// need to break here
	//				BuildAABB(geometries[i], index - primitiveIndex);
	//				break;
	//			}
	//		}
	//	}
	//}	 
}

int main()
{
	PathTracerState state;

	std::vector<Triangle> trianglesInMesh;
	LoadMesh(R"(..\..\sceneResources\sphere.obj)", trianglesInMesh);
	Geometry* triangleMeshGeometry = new Geometry(GeometryType::TRIANGLEMESH, glm::vec3(0.f, -0.5f, 0.f), glm::vec3(0.0f, 180.0f, 0.0f), glm::vec3(1.5f), trianglesInMesh);

	Geometry* topPlaneLightGeometry = new Geometry(GeometryType::PLANE, glm::vec3(0.f, 7.4f, 0.f), glm::vec3(90.f, 0.f, 0.f), glm::vec3(5.f));
	Geometry* leftPlaneLightGeometry = new Geometry(GeometryType::PLANE, glm::vec3(-5.f, 0.f, 0.f), glm::vec3(0.f, 90.f, 0.f), glm::vec3(5.f));
	Geometry* bottomPlaneWhiteGeometry = new Geometry(GeometryType::PLANE, glm::vec3(0.f, -7.5f, 0.f), glm::vec3(90.f, 0.f, 0.f), glm::vec3(15.f));
	Geometry* topPlaneWhiteGeometry = new Geometry(GeometryType::PLANE, glm::vec3(0.f, 7.5f, 0.f), glm::vec3(90.f, 0.f, 0.f), glm::vec3(15.f));
	Geometry* backPlaneWhiteGeometry = new Geometry(GeometryType::PLANE, glm::vec3(0.f, 0.f, -7.5f), glm::vec3(0.f), glm::vec3(15.f));
	Geometry* leftPlaneRedGeometry = new Geometry(GeometryType::PLANE, glm::vec3(-7.5f, 0.f, 0.f), glm::vec3(0.f, 90.f, 0.f), glm::vec3(15.f));
	Geometry* rightPlaneGreenGeometry = new Geometry(GeometryType::PLANE, glm::vec3(7.5f, 0.f, 0.f), glm::vec3(0.f, 90.f, 0.f), glm::vec3(15.f));


	BXDF* diffusebxdfREDMesh = new BXDF();
	diffusebxdfREDMesh->m_type = BXDFTyp::DIFFUSE;
	diffusebxdfREDMesh->m_albedo = { 1.f, 0.f, 0.f };

	BXDF* diffusebxdfGREENMesh = new BXDF();
	diffusebxdfGREENMesh->m_type = BXDFTyp::DIFFUSE;
	diffusebxdfGREENMesh->m_albedo = { 0.f, 1.f, 0.f };

	BXDF* diffusebxdfBLUEMesh = new BXDF();
	diffusebxdfBLUEMesh->m_type = BXDFTyp::DIFFUSE;
	diffusebxdfBLUEMesh->m_albedo = { 0.f, 0.f, 1.f };

	BXDF* diffusebxdfPURPLEMesh = new BXDF();
	diffusebxdfPURPLEMesh->m_type = BXDFTyp::DIFFUSE;
	diffusebxdfPURPLEMesh->m_albedo = { 1.f, 0.f, 1.f };

	BXDF* diffusebxdfWHITEMesh = new BXDF();
	diffusebxdfWHITEMesh->m_type = BXDFTyp::DIFFUSE;
	diffusebxdfWHITEMesh->m_albedo = { 1.f, 1.f, 1.f };

	BXDF* lightbxdfPlane = new BXDF();
	lightbxdfPlane->m_type = BXDFTyp::EMITTER;
	lightbxdfPlane->m_intensity = 5.0f;
	lightbxdfPlane->m_emissiveColor = { 1.f, 1.f, 1.f };

	triangleMeshGeometry->m_bxdf = diffusebxdfWHITEMesh;
	bottomPlaneWhiteGeometry->m_bxdf = diffusebxdfWHITEMesh;
	backPlaneWhiteGeometry->m_bxdf = diffusebxdfWHITEMesh;
	topPlaneWhiteGeometry->m_bxdf = diffusebxdfWHITEMesh;
	leftPlaneRedGeometry->m_bxdf = diffusebxdfREDMesh;
	rightPlaneGreenGeometry->m_bxdf = diffusebxdfGREENMesh;
	topPlaneLightGeometry->m_bxdf = lightbxdfPlane;
	leftPlaneLightGeometry->m_bxdf = lightbxdfPlane;
	
	std::vector<Geometry> geometries;
	geometries.push_back(*triangleMeshGeometry);
	geometries.push_back(*topPlaneLightGeometry);
	//geometries.push_back(*leftPlaneLightGeometry);
	geometries.push_back(*bottomPlaneWhiteGeometry);
	geometries.push_back(*backPlaneWhiteGeometry);
	geometries.push_back(*topPlaneWhiteGeometry);
	geometries.push_back(*rightPlaneGreenGeometry);
	geometries.push_back(*leftPlaneRedGeometry);

	// TODO: Load scene from file
	int windowWidth  = 800;
	int windowHeight = 800;
	int cameraResolution = windowWidth * windowHeight;

	// First we will copy the base geometry object to device memory
	state.d_geometry = nullptr;
	hipMalloc((void**)&(state.d_geometry), sizeof(Geometry) * geometries.size());
	cudaCheckErrors("hipMalloc geometry fail");
	hipMemcpy(state.d_geometry, geometries.data(), sizeof(Geometry) * geometries.size(), hipMemcpyHostToDevice);
	cudaCheckErrors("hipMemcpy geometry fail");
	state.d_raytracableObjects = geometries.size();

	// Create and load AABB
	state.d_AABB = nullptr;
	hipMalloc((void**)&(state.d_AABB), sizeof(AABB) * GetTotalPrimitiveCount(geometries));
	cudaCheckErrors("hipMalloc geometry fail");
	int blockSize = 256;
	int numBlocks = (GetTotalPrimitiveCount(geometries) + 1) / blockSize;
	CreateWorldAABB << <numBlocks, blockSize >> > (state.d_geometry, geometries.size(), GetTotalPrimitiveCount(geometries), state.d_AABB);
	// Get the memory back from the GPU to CPU
	std::vector<AABB> aabbArray; 
	aabbArray.resize(GetTotalPrimitiveCount(geometries));
	hipMemcpy(aabbArray.data(), state.d_AABB, GetTotalPrimitiveCount(geometries), hipMemcpyDeviceToHost);
	//Create K-D Tree
	std::shared_ptr<KDTree> kdTree = std::make_shared<KDTree>(aabbArray, 0, 2);


	// Now we will save the internal triangle data to device memory
	for (int i = 0; i < geometries.size(); ++i)
	{
		BXDF* hostBXDFData;
		hipMallocManaged((void**)&hostBXDFData, sizeof(BXDF));
		cudaCheckErrors("hipMalloc host bxdf data fail");
		hipMemcpy(hostBXDFData, geometries[i].m_bxdf, sizeof(BXDF), hipMemcpyHostToDevice);
		cudaCheckErrors("hipMemcpy host bxdf data fail");
		hipMemcpy(&(state.d_geometry[i].m_bxdf), &hostBXDFData, sizeof(BXDF*), hipMemcpyHostToDevice);
		cudaCheckErrors("hipMemcpy device bxdf data fail");

		if (geometries[i].m_geometryType == GeometryType::TRIANGLEMESH)
		{
			// TODO: Figure out a better way to allocate and deallocate this hostTriangleData
			Triangle* hostTriangleData;
			hipMallocManaged((void**)&hostTriangleData, sizeof(Triangle) * geometries[i].m_numberOfTriangles);
			cudaCheckErrors("hipMalloc host triangle data fail");
			hipMemcpy(hostTriangleData, geometries[i].m_triangles, sizeof(Triangle) * geometries[i].m_numberOfTriangles, hipMemcpyHostToDevice);
			cudaCheckErrors("hipMemcpy host triangle data fail");
			hipMemcpy(&(state.d_geometry[i].m_triangles), &hostTriangleData, sizeof(Triangle*), hipMemcpyHostToDevice);
			cudaCheckErrors("hipMemcpy device triangle data fail");
		}
	}

	Camera camera;
	camera.m_position = glm::vec3(0.f, 0.f, 15.f);
	camera.m_forward = glm::vec3(0.f, 0.f, -1.f);
	camera.m_worldUp = glm::vec3(0.f, 1.f, 0.f);
	camera.m_fov = 70.f;
	camera.m_screenHeight = float(windowWidth);
	camera.m_screenWidth = float(windowHeight);
	camera.m_nearClip = 0.1f;
	camera.m_farClip = 1000.f;
	camera.m_pitch = 0.f;
	camera.m_yaw = -90.f;
	camera.UpdateBasisAxis();

	camera.m_invViewProj = camera.GetInverseViewMatrix() * camera.GetInverseProjectionMatrix();

	GLFWViewer* viewer = new GLFWViewer(windowWidth, windowHeight);

	glm::vec3* d_pixelColor = nullptr;
	hipMalloc((void**)&(d_pixelColor), sizeof(glm::vec3) * windowWidth * windowHeight);
	cudaCheckErrors("hipMalloc d_pixelColor fail");

	state.d_camera = nullptr;
	hipMalloc((void**)&(state.d_camera), sizeof(Camera));
	cudaCheckErrors("hipMalloc camera fail");

	int iteration = 1;

	int maxDepth = 4;
	int samplesPerPixel = 16;

	GpuTimer timer;
	float time = 0.f;

	while (!glfwWindowShouldClose(viewer->m_window))
	{
		processInput(viewer->m_window, camera, viewer, iteration, time);
		camera.m_invViewProj = camera.GetInverseViewMatrix() * camera.GetInverseProjectionMatrix();

		//
		// EXECUTE CUDA KERNEL ON RENDER BUFFER
		//

		hipGraphicsMapResources(1, &viewer->interop->cgr[viewer->interop->index], viewer->stream);
		{
			timer.Start();
			viewer->cuda_err = pxl_kernel_launcher(viewer->interop->ca[viewer->interop->index] ,
				windowWidth,
				windowHeight,
				viewer->event,
				viewer->stream,
				state.d_geometry, 
				camera, 
				geometries.size(),
				iteration, 
				maxDepth,
				samplesPerPixel,
				d_pixelColor);
			timer.Stop();
		}
		hipGraphicsUnmapResources(1, &viewer->interop->cgr[viewer->interop->index], viewer->stream);

		char title[256];
		time = timer.Elapsed();
		sprintf(title, "Firefly | FPS %f | iteration: %d | kernel took: %.2fs | samples per pixel: %d | max depth: %d", 1.0f/time, iteration, time/iteration, samplesPerPixel, maxDepth);
		glfwSetWindowTitle(viewer->m_window, title);

		//
		// BLIT & SWAP FBO
		// 
		glBlitNamedFramebuffer(viewer->interop->fb[viewer->interop->index], 0,
			0, 0, viewer->interop->width, viewer->interop->height,
			0, viewer->interop->height, viewer->interop->width, 0,
			GL_COLOR_BUFFER_BIT,
			GL_NEAREST);

		viewer->interop->index = (viewer->interop->index + 1) % viewer->interop->count;
		iteration++;

		glfwSwapBuffers(viewer->m_window);
		glfwPollEvents();
	}

	glfwDestroyWindow(viewer->m_window);
	glfwTerminate();

	hipFree(state.d_geometry);
	delete viewer;
	hipFree(d_pixelColor);
	return 0;
}
