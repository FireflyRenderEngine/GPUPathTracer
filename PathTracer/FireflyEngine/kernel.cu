#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "utilities.h"

#include <cmath>

__device__ bool intersectPlane(const Geometry& plane, const Ray& ray, Intersect& intersect)
{
	// CLARIFICATION: all the rays need to be in object space; convert the ray to world space elsewhere
	float denom = glm::dot(plane.m_normal, ray.m_direction);
	if (glm::abs(denom) > 1e-7)
	{
		glm::vec3 p0l0 = -ray.m_origin;
		float t = glm::dot(p0l0, plane.m_normal) / denom;
		glm::vec3 P = ray.m_origin + t * ray.m_direction;
		// check bounds of the plane centered at 0,0,0 in object space
		if (!(P.x >= -0.5f && P.x <= 0.5f && P.y >= -0.5f && P.y <= 0.5f))
		{
			return false;
		}
		intersect.m_t = t;
		intersect.m_intersectionPoint = P;
		intersect.m_normal = plane.m_normal;
	
		return (intersect.m_t > 0);
	}
	return false;
}

// fast Triangle intersection : https://cadxfem.org/inf/Fast%20MinimumStorage%20RayTriangle%20Intersection.pdf
__device__ bool intersectTriangle(const Triangle& triangle, const Ray& ray, Intersect& intersect)
{
	// CLARIFICATION: all the rays need to be in object space; convert the ray to world space elsewhere
	const float EPSILON = 0.000001;
	glm::vec3 vertex0 = triangle.m_v0;
	glm::vec3 vertex1 = triangle.m_v1;
	glm::vec3 vertex2 = triangle.m_v2;
	glm::vec3 edge1, edge2, pvec, tvec, qvec;
	float det, invDet, u, v;
	edge1 = vertex1 - vertex0;
	edge2 = vertex2 - vertex0;

	// Normal for backface culling
	glm::vec3 Normal = glm::cross(edge1, edge2);
	if (glm::dot(ray.m_direction, Normal) > 0) {
		return false; // back-facing surface
	}

	pvec = glm::cross(ray.m_direction, edge2);
	det = glm::dot(edge1, pvec);

	// BACKFACE CULLING
	if (det < EPSILON) {
		return false;    // This ray is parallel to this triangle.
	}

	tvec = ray.m_origin - vertex0;
	u = glm::dot(tvec, pvec);

	if (u < 0.0f || u > det) {
		return false;
	}

	qvec = glm::cross(tvec, edge1);

	v = glm::dot(ray.m_direction, qvec);
	if (v < 0.0f || u + v > det) {
		return false;
	}

	float t = glm::dot(edge2, qvec);

	invDet = 1.0 / det;

	t *= invDet;
	u *= invDet;
	v *= invDet;

	//if (det)

	////u = invDet * glm::dot(tvec, pvec);
	//if (u < 0.0 || u > 1.0)
	//{
	//	return false;
	//}
	////v = invDet * ;
	//if (v < 0.0 || u + v > 1.0)
	//	return false;


	// At this stage we can compute t to find out where the intersection point is on the line.
	//float t = invDet * glm::dot(edge2, qvec);
	if (t > EPSILON) // ray intersection
	{
		intersect.m_intersectionPoint = ray.m_origin + ray.m_direction * t;
		intersect.m_t = t;
		intersect.m_normal = glm::normalize(glm::cross(edge1, edge2));
		return true;
	}
	else // This means that there is a line intersection but not a ray intersection.
	{
		return false;
	}
}

__device__ bool setIntersection(double& tMax, Intersect& intersect, Intersect& objectSpaceIntersect, glm::mat4 modelMatrix, const Ray& ray)
{
	// convert point of intersection into world space
	glm::vec3 worldPOI = modelMatrix * glm::vec4(intersect.m_intersectionPoint, 1.0f);
	double distanceOfPOI = glm::distance(worldPOI, ray.m_origin);
	if (distanceOfPOI < tMax)
	{
		intersect.m_normal = glm::inverse(glm::transpose(modelMatrix)) * glm::vec4(objectSpaceIntersect.m_normal, 0.f);
		intersect.m_intersectionPoint = worldPOI;
		intersect.m_t = distanceOfPOI;
		intersect.m_hit = true;
		tMax = distanceOfPOI;
		return true;
	}
	return false;
}

__device__ Intersect intersectRays(const Ray& ray, Geometry* geometries, unsigned int raytracableObjects)
{
	// This is the global intersect that stores the intersect info in world space
	Intersect intersect;

	// loop through all geometries, find the smallest "t" value for a single ray
	for (int i = 0; i < raytracableObjects; ++i)
	{
		Geometry& geometry = geometries[i];

		// Generate the ray in the object space of the geometry being intersected.
		Ray& objectSpaceRay = Ray(geometry.m_inverseModelMatrix * glm::vec4(ray.m_origin, 1.f), glm::normalize(geometry.m_inverseModelMatrix * glm::vec4(ray.m_direction, 0.f)));

		double tMax = INFINITY;
		// This intersect is re-created each iteration and stores the intersect info in object space of the geometry
		Intersect objectSpaceIntersect;

		if (geometry.m_geometryType == GeometryType::TRIANGLEMESH)
		{
			for (int j = 0; j < geometry.m_numberOfTriangles; ++j)
			{

				if (intersectTriangle(geometry.m_triangles[j], objectSpaceRay, objectSpaceIntersect))
				{
					if (setIntersection(tMax, intersect, objectSpaceIntersect, geometry.m_modelMatrix, ray)) {
						intersect.geometryIndex = i;
						intersect.triangleIndex = j;
					}
				}
			}
		}
		else if (geometry.m_geometryType == GeometryType::PLANE)
		{
			if (intersectPlane(geometry, objectSpaceRay, intersect))
			{
				if (setIntersection(tMax, intersect, objectSpaceIntersect, geometry.m_modelMatrix, ray)) {
					intersect.geometryIndex = i;
				}
			}
		}
		else if (geometry.m_geometryType == GeometryType::SPHERE)
		{
			printf("Sphere Geometry implemented yet!");
		}
		else
		{
			printf("No such Geometry implemented yet!");
		}
	}
	return intersect;
}

__device__ glm::vec3 shade(const Ray& incomingRay, const Intersect& intersect, glm::vec3& outgoingRayDirection, Geometry* geometries)
{
	Geometry hitGeometry = geometries[intersect.geometryIndex];

	Ray& objectSpaceRay = Ray(hitGeometry.m_inverseModelMatrix * glm::vec4(incomingRay.m_origin, 1.f), hitGeometry.m_inverseModelMatrix * glm::vec4(incomingRay.m_direction, 0.f));
	return glm::abs(intersect.m_normal);// hitGeometry.m_bxdf->bsdf(-objectSpaceRay.m_direction, intersect.m_normal, outgoingRayDirection, intersect);
}

__device__ void generateRays(uchar3* pbo, Camera camera, Geometry* geometries, unsigned int raytracableObjects)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int pixelSize = camera.m_screenHeight * camera.m_screenWidth;
	int pixelIndex = y * camera.m_screenWidth + x;

	if (pixelIndex >= pixelSize)
	{
		return;
	}
	Ray ray;
	ray.m_origin = camera.m_position;

	float Px = (x / camera.m_screenWidth) * 2.f - 1.f;
	float Py = 1.f - (y / camera.m_screenHeight) * 2.f;

	glm::vec3 wLookAtPoint = camera.GetInverseViewMatrix() * camera.GetInverseProjectionMatrix() * (glm::vec4(Px, Py, 1.f, 1.f) * camera.m_farClip);

	ray.m_direction = glm::normalize(wLookAtPoint - ray.m_origin);

	Intersect intersect = intersectRays(ray, geometries, raytracableObjects);

	if (intersect.m_hit)
	{
		Ray outgoingRay;
		outgoingRay.m_origin = intersect.m_intersectionPoint;
		glm::vec3 pixelColor = shade(ray, intersect, outgoingRay.m_direction, geometries);
		pbo[pixelIndex] = make_uchar3(pixelColor.x*255.f, pixelColor.y*255.f, pixelColor.z*255.f);
	}
}

__global__ void launchPathTrace(uchar3* pbo, PathTracerState* state, Camera camera)
{
	generateRays(pbo, camera, state->d_geometry, state->d_raytracableObjects);
}

int main()
{
	PathTracerState* state;

	hipMallocManaged((void**)&state, sizeof(PathTracerState));

	std::vector<Triangle> trianglesInMesh;
	LoadMesh(R"(..\..\sceneResources\rocketman.obj)", trianglesInMesh);
	Geometry* triangleMeshGeometry = new Geometry(GeometryType::TRIANGLEMESH, glm::vec3(0), glm::vec3(0.0f, 90.0f, 180.0f), glm::vec3(1.0f), trianglesInMesh);

	Geometry*  planeLightGeometry = new Geometry(GeometryType::PLANE, glm::vec3(0.f, 0.f, 2.5f), glm::vec3(0.f, 0.f, 0.f), glm::vec3(5.f));
	Geometry* planeLightGeometry1 = new Geometry(GeometryType::PLANE, glm::vec3(0.f, 0.f, -2.5f), glm::vec3(0.f, 0.f, 0.f), glm::vec3(5.f));
	Geometry* planeLightGeometry2 = new Geometry(GeometryType::PLANE, glm::vec3(0.f, -2.5f, 0.f), glm::vec3(90.f, 0.f, 0.f), glm::vec3(5.f));
	Geometry* planeLightGeometry3 = new Geometry(GeometryType::PLANE, glm::vec3(0.f, 2.5f, 0.f), glm::vec3(90.f, 0.f, 0.f), glm::vec3(5.f));


	BXDF* diffusebxdfMesh = new BXDF();
	diffusebxdfMesh->m_type = BXDFTyp::DIFFUSE;
	diffusebxdfMesh->m_albedo = { 1.f, 0.f, 0.f };

	BXDF* lightbxdfPlane = new BXDF();
	lightbxdfPlane->m_type = BXDFTyp::EMITTER;
	lightbxdfPlane->m_intensity = 2.0f;
	lightbxdfPlane->m_emissiveColor = { 1.f, 1.f, 1.f };

	triangleMeshGeometry->m_bxdf = diffusebxdfMesh;
	planeLightGeometry->m_bxdf = diffusebxdfMesh;
	planeLightGeometry1->m_bxdf = diffusebxdfMesh;
	planeLightGeometry2->m_bxdf = diffusebxdfMesh;
	planeLightGeometry3->m_bxdf = diffusebxdfMesh;

	std::vector<Geometry> geometries;
	//geometries.push_back(*triangleMeshGeometry);
	geometries.push_back(*planeLightGeometry);
	geometries.push_back(*planeLightGeometry1);
	geometries.push_back(*planeLightGeometry2);
	geometries.push_back(*planeLightGeometry3);

	// TODO: Load scene from file
	int windowWidth  = 800;
	int windowHeight = 800;
	int cameraResolution = windowWidth * windowHeight;

	int samplesPerPixel = 1;

	// First we will copy the base geometry object to device memory
	state->d_geometry = nullptr;
	hipMalloc((void**)&(state->d_geometry), sizeof(Geometry) * geometries.size());
	cudaCheckErrors("hipMalloc geometry fail");
	hipMemcpy(state->d_geometry, geometries.data(), sizeof(Geometry) * geometries.size(), hipMemcpyHostToDevice);
	cudaCheckErrors("hipMemcpy geometry fail");
	state->d_raytracableObjects = geometries.size();

	// Now we will save the internal triangle data to device memory
	for (int i = 0; i < geometries.size(); ++i)
	{
		BXDF* hostBXDFData;
		hipMallocManaged((void**)&hostBXDFData, sizeof(BXDF));
		cudaCheckErrors("hipMalloc host bxdf data fail");
		hipMemcpy(hostBXDFData, geometries[i].m_bxdf, sizeof(BXDF), hipMemcpyHostToDevice);
		cudaCheckErrors("hipMemcpy host bxdf data fail");
		hipMemcpy(&(state->d_geometry[i].m_bxdf), &hostBXDFData, sizeof(BXDF*), hipMemcpyHostToDevice);
		cudaCheckErrors("hipMemcpy device bxdf data fail");

		if (geometries[i].m_geometryType == GeometryType::TRIANGLEMESH)
		{
			// TODO: Figure out a better way to allocate and deallocate this hostTriangleData
			Triangle* hostTriangleData;
			hipMallocManaged((void**)&hostTriangleData, sizeof(Triangle) * geometries[i].m_numberOfTriangles);
			cudaCheckErrors("hipMalloc host triangle data fail");
			hipMemcpy(hostTriangleData, geometries[i].m_triangles, sizeof(Triangle) * geometries[i].m_numberOfTriangles, hipMemcpyHostToDevice);
			cudaCheckErrors("hipMemcpy host triangle data fail");
			hipMemcpy(&(state->d_geometry[i].m_triangles), &hostTriangleData, sizeof(Triangle*), hipMemcpyHostToDevice);
			cudaCheckErrors("hipMemcpy device triangle data fail");
		}
	}

	state->d_raysToTrace = 0;
	hipMalloc((void**)&(state->d_raysToTrace), cameraResolution * samplesPerPixel * sizeof(unsigned int));
	cudaCheckErrors("hipMalloc rays fail");

	glm::vec3* pixels = new glm::vec3[cameraResolution];

	state->d_pixels = nullptr;
	hipMalloc((void**)&(state->d_pixels), cameraResolution * sizeof(glm::vec3));
	cudaCheckErrors("hipMalloc pixels fail");
	hipMemcpy(state->d_pixels, pixels, cameraResolution * sizeof(glm::vec3), hipMemcpyHostToDevice);
	cudaCheckErrors("hipMemcpy pixels fail");

	dim3 blockSize(16, 16, 1);
	dim3 gridSize;
	gridSize.x = (windowWidth / blockSize.x);// +1;
	gridSize.y = (windowHeight / blockSize.y);// +1;

	Camera camera;
	camera.m_position = glm::vec3(0.f, 0.f, 15.f);
	camera.m_forward = glm::vec3(0.f, 0.f, -1.f);
	camera.m_worldUp = glm::vec3(0.f, 1.f, 0.f);
	camera.m_fov = 70.f;
	camera.m_screenHeight = float(windowWidth);
	camera.m_screenWidth = float(windowHeight);
	camera.m_nearClip = 0.1f;
	camera.m_farClip = 1000.f;
	camera.m_pitch = 0.f;
	camera.m_yaw = -90.f;
	camera.UpdateBasisAxis();

	GLFWViewer* viewer = new GLFWViewer(windowWidth, windowHeight, pixels);
	viewer->Create();

	state->d_camera = nullptr;
	hipMalloc((void**)&(state->d_camera), sizeof(Camera));
	cudaCheckErrors("hipMalloc camera fail");

	while (!glfwWindowShouldClose(viewer->m_window))
	{
		processInput(viewer->m_window, camera, pixels);

		uchar3* pbo_dptr = NULL;
		size_t num_bytes;
		hipGraphicsResource* pboResource = (viewer->getPBOResource());
		hipGraphicsMapResources(1, &pboResource, 0);
		hipGraphicsResourceGetMappedPointer((void**)&pbo_dptr, &num_bytes, pboResource);
		hipMemset(pbo_dptr, 0, num_bytes);
		{
			launchPathTrace << < gridSize, blockSize >> > (pbo_dptr, state, camera);
		}
		hipGraphicsUnmapResources(1, &pboResource, 0);

		std::string title = "Firefly";
		glfwSetWindowTitle(viewer->m_window, title.c_str());

		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, viewer->getPBO());
		glBindTexture(GL_TEXTURE_2D, viewer->getTexture());
		glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, windowWidth, windowHeight, GL_RGB, GL_UNSIGNED_BYTE, NULL);

		glClearColor(0.0f, 0.f, 0.f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);

		viewer->Draw();
		glfwSwapBuffers(viewer->m_window);
		glfwPollEvents();
	}
	
	cleanCUDAMemory(state);
	delete[] pixels;
	delete viewer;
	delete triangleMeshGeometry;
	//hipFree(hostTriangleData);
	return 0;
}
