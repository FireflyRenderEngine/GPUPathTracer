#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "utilities.h"

#include <cmath>

surface<void, cudaSurfaceType2D> surf;

__device__ bool intersectPlane(const Geometry& plane, const Ray& ray, Intersect& intersect)
{
	// CLARIFICATION: all the rays need to be in object space; convert the ray to world space elsewhere
	float denom = glm::dot(plane.m_normal, ray.m_direction);
	if (glm::abs(denom) > 1e-7)
	{
		glm::vec3 p0l0 = -ray.m_origin;
		float t = glm::dot(p0l0, plane.m_normal) / denom;
		glm::vec3 P = ray.m_origin + t * ray.m_direction;
		// check bounds of the plane centered at 0,0,0 in object space
		if (!(P.x >= -0.5f && P.x <= 0.5f && P.y >= -0.5f && P.y <= 0.5f))
		{
			return false;
		}
	
		if (t > 0.0f) {
			intersect.m_t = t;
			intersect.m_intersectionPoint = P;
			intersect.m_normal = plane.m_normal;
			return true;
		}
		return false;
	}
	return false;
}

// fast Triangle intersection : https://cadxfem.org/inf/Fast%20MinimumStorage%20RayTriangle%20Intersection.pdf
__device__ bool intersectTriangle(const Triangle& triangle, const Ray& ray, Intersect& intersect)
{
	// CLARIFICATION: all the rays need to be in object space; convert the ray to world space elsewhere
	const float EPSILON = 0.000001;
	glm::vec3 vertex0 = triangle.m_v0;
	glm::vec3 vertex1 = triangle.m_v1;
	glm::vec3 vertex2 = triangle.m_v2;
	glm::vec3 edge1, edge2, pvec, tvec, qvec;
	float det, invDet, u, v;
	edge1 = vertex1 - vertex0;
	edge2 = vertex2 - vertex0;

	pvec = glm::cross(ray.m_direction, edge2);
	det = glm::dot(edge1, pvec);

	// BACKFACE CULLING
	if (det < EPSILON) {
		return false;    // This ray is parallel to this triangle.
	}

	tvec = ray.m_origin - vertex0;
	u = glm::dot(tvec, pvec);

	if (u < 0.0f || u > det) {
		return false;
	}

	qvec = glm::cross(tvec, edge1);

	v = glm::dot(ray.m_direction, qvec);
	if (v < 0.0f || u + v > det) {
		return false;
	}

	float t = glm::dot(edge2, qvec);

	invDet = 1.0 / det;

	t *= invDet;
	u *= invDet;
	v *= invDet;

	if (t > EPSILON) // ray intersection
	{
		glm::vec3 intersectPoint = ray.m_origin + ray.m_direction * t;
		intersect.m_intersectionPoint = intersectPoint;
		intersect.m_t = t;

		// Calculate the normal using barycentric coordinates
		float denom = (vertex1.y - vertex2.y) * (vertex0.x - vertex2.x) + (vertex2.x - vertex1.x) * (vertex0.y - vertex2.y);
		float wv1 = ((vertex1.y - vertex2.y) * (intersectPoint.x - vertex2.x) + (vertex2.x - vertex1.x) * (intersectPoint.y - vertex2.y)) / denom;
		float wv2 = ((vertex2.y - vertex0.y) * (intersectPoint.x - vertex2.x) + (vertex0.x - vertex2.x) * (intersectPoint.y - vertex2.y)) / denom;
		float wv3 = 1 - wv1 - wv2;
		intersect.m_normal = glm::normalize((wv1 * triangle.m_n0) + (wv2 * triangle.m_n1) + (wv3 * triangle.m_n2));
		return true;
	}
	else // This means that there is a line intersection but not a ray intersection.
	{
		return false;
	}
}

__device__ bool setIntersection(float& tMax, Intersect& intersectOut, const Intersect& objectSpaceIntersect, glm::mat4 invTransModelMatrix, glm::mat4 modelMatrix,const Ray& ray)
{
	// convert point of intersection into world space
	glm::vec3 worldPOI = modelMatrix * glm::vec4(objectSpaceIntersect.m_intersectionPoint, 1.0f);
	float distanceOfPOI = glm::distance(worldPOI, ray.m_origin);
	if (distanceOfPOI < tMax)
	{
		intersectOut.m_normal = glm::normalize(glm::vec3(invTransModelMatrix * glm::vec4(objectSpaceIntersect.m_normal, 0.f)));
		intersectOut.m_intersectionPoint = worldPOI;
		intersectOut.m_t = distanceOfPOI;
		intersectOut.m_hit = true;
		tMax = distanceOfPOI;
		return true;
	}
	return false;
}

__device__ Intersect& intersectRays(const Ray& ray, Geometry* geometries, unsigned int raytracableObjects)
{
	// This is the global intersect that stores the intersect info in world space
	Intersect intersectOut;
	float tMax = INFINITY;
	// loop through all geometries, find the smallest "t" value for a single ray
	for (int i = 0; i < raytracableObjects; ++i)
	{
		Geometry& geometry = geometries[i];

		// Generate the ray in the object space of the geometry being intersected.
		Ray& objectSpaceRay = Ray(geometry.m_inverseModelMatrix * glm::vec4(ray.m_origin, 1.f), glm::normalize(geometry.m_inverseModelMatrix * glm::vec4(ray.m_direction, 0.f)));

		// This intersect is re-created each iteration and stores the intersect info in object space of the geometry
		Intersect objectSpaceIntersect;

		if (geometry.m_geometryType == GeometryType::TRIANGLEMESH)
		{
			for (int j = 0; j < geometry.m_numberOfTriangles; ++j)
			{

				if (intersectTriangle(geometry.m_triangles[j], objectSpaceRay, objectSpaceIntersect))
				{
					if (setIntersection(tMax, intersectOut, objectSpaceIntersect, geometry.m_invTransModelMatrix, geometry.m_modelMatrix, ray)) {
						intersectOut.geometryIndex = i;
						intersectOut.triangleIndex = j;
					}
				}
			}
		}
		else if (geometry.m_geometryType == GeometryType::PLANE)
		{
			if (intersectPlane(geometry, objectSpaceRay, objectSpaceIntersect))
			{
				if (setIntersection(tMax, intersectOut, objectSpaceIntersect, geometry.m_invTransModelMatrix, geometry.m_modelMatrix, ray)) {
					intersectOut.geometryIndex = i;
				}
			}
		}
		else if (geometry.m_geometryType == GeometryType::SPHERE)
		{
			printf("Sphere Geometry implemented yet!");
		}
		else
		{
			printf("No such Geometry implemented yet!");
		}
	}
	return intersectOut;
}

__device__ glm::vec3 getBXDF(const Ray& incomingRay, const Intersect& intersect, glm::vec3& outgoingRayDirection, Geometry* geometries)
{
	return (geometries[intersect.geometryIndex].m_bxdf->bsdf((-incomingRay.m_direction), outgoingRayDirection, intersect));
}

__device__ float getPDF(const Ray& incomingRay, const glm::vec3& outgoingRayDirection, const Intersect& intersect, Geometry* geometries)
{
	return (geometries[intersect.geometryIndex].m_bxdf->pdf((-incomingRay.m_direction), outgoingRayDirection, intersect.m_normal));
}

__device__ Ray& generateRay(Camera camera, int x, int y)
{
	Ray ray;

	// TODO: add stratified samples
	// TODO: add depth of field
	ray.m_origin = camera.m_position;

	float Px = (x / camera.m_screenWidth) * 2.f - 1.f;
	float Py = 1.f - (y / camera.m_screenHeight) * 2.f;

	glm::vec3 wLookAtPoint = camera.m_invViewProj * (glm::vec4(Px, Py, 1.f, 1.f) * camera.m_farClip);

	ray.m_direction = glm::normalize(wLookAtPoint - ray.m_origin);
	return ray;
}

__global__ void launchPathTrace(Geometry* geometries, Camera camera, int numberOfGeometries, int maxIterations)
{
	int x = blockIdx.x* blockDim.x + threadIdx.x;
	int y = blockIdx.y* blockDim.y + threadIdx.y;

	int pixelSize = camera.m_screenHeight * camera.m_screenWidth;
	int pixelIndex = y * camera.m_screenWidth + x;

	if (pixelIndex >= pixelSize)
	{
		return;
	}
	// Do Light transport here
	// Loop until we hit max rays or russian roulette termination
	// 1. Check if we hit a light
	//   1.a if we hit light, then terminate
	// 2. Check what material we hit
	//   2.a get bsdf and pdf
	//   2.b get outgoing ray
	//   2.c calculate thruput and calculate russian roulette
	int iterations = 0;
	glm::vec3 pixelColor(0.f, 0.f, 0.f);
	Ray& ray = generateRay(camera, x, y);

	glm::vec3 thruput(1.f);

	do
	{
		Intersect intersect = intersectRays(ray, geometries, numberOfGeometries);
		if (!intersect.m_hit)
		{
			pixelColor = glm::vec3(0.1, 0.4, 0.2); //REMOVE ME
			thruput *= 0.0f;
			break;
		}
		else {
			Ray outgoingRay;
			outgoingRay.m_origin = intersect.m_intersectionPoint;

			glm::vec3 bxdf = getBXDF(ray, intersect, outgoingRay.m_direction, geometries);
			if (geometries[intersect.geometryIndex].m_bxdf->m_type == BXDFTyp::EMITTER)
			{
				// add to thruput and exit since we hit an emitter
				pixelColor += thruput * bxdf;// do abscos
				thruput *= 0.0f;
				break;
			}

			float pdf = getPDF(ray, outgoingRay.m_direction, intersect, geometries);

			
			// pixelColor += emitted light + integral of (bxdf/pdf)
			if (pdf > 0.001) {
				float dotProd = glm::abs(glm::dot(-glm::normalize(outgoingRay.m_direction), intersect.m_normal));
				printf("dotProd : %f\n", dotProd);
				thruput *= glm::abs(glm::dot(-glm::normalize(outgoingRay.m_direction), intersect.m_normal)) * (bxdf / pdf);
			}

			// set the next ray for iteration
			outgoingRay.m_origin += 0.01f * intersect.m_normal;
			ray = outgoingRay;

		}

		iterations++;
	} while (iterations < maxIterations);

	pixelColor += thruput;
	if (iterations != 0) {
		pixelColor /= iterations;
	}

	surf2Dwrite(make_uchar4(pixelColor[0] * 255.f, pixelColor[1] * 255.f, pixelColor[2] * 255.f, 255.f),
		surf,
		x * sizeof(uchar4),
		y,
		hipBoundaryModeZero);
}

hipError_t pxl_kernel_launcher(hipArray_const_t array,
	const int         width,
	const int         height,
	hipEvent_t       event,
	hipStream_t      stream,
	Geometry* geom,
	Camera camera,
	int numGeom,
	int maxIterations)
{
	hipError_t cuda_err;

	// cuda_err = hipEventRecord(event,stream);

	cuda_err = cudaBindSurfaceToArray(surf, array);

	if (cuda_err)
		return cuda_err;

	dim3 blockSize(16, 16, 1);
	dim3 gridSize;
	gridSize.x = ((width + blockSize.x - 1) / blockSize.x);
	gridSize.y = ((height + blockSize.y -1) / blockSize.y);
	const int blocks = (width * height + 256 - 1) / 256;
	
	launchPathTrace << <gridSize, blockSize , 0, stream >> > (geom, camera, numGeom, maxIterations);

	return hipSuccess;
}

int main()
{
	PathTracerState state;

	std::vector<Triangle> trianglesInMesh;
	LoadMesh(R"(..\..\sceneResources\sphere.obj)", trianglesInMesh);
	Geometry* triangleMeshGeometry = new Geometry(GeometryType::TRIANGLEMESH, glm::vec3(0), glm::vec3(0.0f, 180.0f, 0.0f), glm::vec3(1.0f), trianglesInMesh);

	Geometry*  topPlaneLightGeometry = new Geometry(GeometryType::PLANE, glm::vec3(0.f, 5.f, 0.f), glm::vec3(90.f, 0.f, 0.f), glm::vec3(5.f));
	Geometry* leftPlaneLightGeometry = new Geometry(GeometryType::PLANE, glm::vec3(-5.f, 0.f, 0.f), glm::vec3(0.f, 90.f, 0.f), glm::vec3(5.f));

	BXDF* diffusebxdfREDMesh = new BXDF();
	diffusebxdfREDMesh->m_type = BXDFTyp::DIFFUSE;
	diffusebxdfREDMesh->m_albedo = { 1.f, 0.f, 0.f };

	BXDF* diffusebxdGREENfMesh = new BXDF();
	diffusebxdGREENfMesh->m_type = BXDFTyp::DIFFUSE;
	diffusebxdGREENfMesh->m_albedo = { 0.f, 1.f, 0.f };

	BXDF* diffusebxdfBLUEMesh = new BXDF();
	diffusebxdfBLUEMesh->m_type = BXDFTyp::DIFFUSE;
	diffusebxdfBLUEMesh->m_albedo = { 0.f, 0.f, 1.f };

	BXDF* diffusebxdfPURPLEMesh = new BXDF();
	diffusebxdfPURPLEMesh->m_type = BXDFTyp::DIFFUSE;
	diffusebxdfPURPLEMesh->m_albedo = { 1.f, 0.f, 1.f };

	BXDF* lightbxdfPlane = new BXDF();
	lightbxdfPlane->m_type = BXDFTyp::EMITTER;
	lightbxdfPlane->m_intensity = 2.0f;
	lightbxdfPlane->m_emissiveColor = { 1.f, 1.f, 1.f };

	triangleMeshGeometry->m_bxdf = diffusebxdfREDMesh;
	topPlaneLightGeometry->m_bxdf = lightbxdfPlane;
	leftPlaneLightGeometry->m_bxdf = lightbxdfPlane;

	std::vector<Geometry> geometries;
	geometries.push_back(*triangleMeshGeometry);
	geometries.push_back(*topPlaneLightGeometry);
	geometries.push_back(*leftPlaneLightGeometry);

	// TODO: Load scene from file
	int windowWidth  = 800;
	int windowHeight = 800;
	int cameraResolution = windowWidth * windowHeight;

	int samplesPerPixel = 1;

	// First we will copy the base geometry object to device memory
	state.d_geometry = nullptr;
	hipMalloc((void**)&(state.d_geometry), sizeof(Geometry) * geometries.size());
	cudaCheckErrors("hipMalloc geometry fail");
	hipMemcpy(state.d_geometry, geometries.data(), sizeof(Geometry) * geometries.size(), hipMemcpyHostToDevice);
	cudaCheckErrors("hipMemcpy geometry fail");
	state.d_raytracableObjects = geometries.size();

	// Now we will save the internal triangle data to device memory
	for (int i = 0; i < geometries.size(); ++i)
	{
		BXDF* hostBXDFData;
		hipMallocManaged((void**)&hostBXDFData, sizeof(BXDF));
		cudaCheckErrors("hipMalloc host bxdf data fail");
		hipMemcpy(hostBXDFData, geometries[i].m_bxdf, sizeof(BXDF), hipMemcpyHostToDevice);
		cudaCheckErrors("hipMemcpy host bxdf data fail");
		hipMemcpy(&(state.d_geometry[i].m_bxdf), &hostBXDFData, sizeof(BXDF*), hipMemcpyHostToDevice);
		cudaCheckErrors("hipMemcpy device bxdf data fail");

		if (geometries[i].m_geometryType == GeometryType::TRIANGLEMESH)
		{
			// TODO: Figure out a better way to allocate and deallocate this hostTriangleData
			Triangle* hostTriangleData;
			hipMallocManaged((void**)&hostTriangleData, sizeof(Triangle) * geometries[i].m_numberOfTriangles);
			cudaCheckErrors("hipMalloc host triangle data fail");
			hipMemcpy(hostTriangleData, geometries[i].m_triangles, sizeof(Triangle) * geometries[i].m_numberOfTriangles, hipMemcpyHostToDevice);
			cudaCheckErrors("hipMemcpy host triangle data fail");
			hipMemcpy(&(state.d_geometry[i].m_triangles), &hostTriangleData, sizeof(Triangle*), hipMemcpyHostToDevice);
			cudaCheckErrors("hipMemcpy device triangle data fail");
		}
	}

	state.d_raysToTrace = 0;
	hipMalloc((void**)&(state.d_raysToTrace), cameraResolution * samplesPerPixel * sizeof(unsigned int));
	cudaCheckErrors("hipMalloc rays fail");

	glm::vec3* pixels = new glm::vec3[cameraResolution];

	Camera camera;
	camera.m_position = glm::vec3(0.f, 0.f, 15.f);
	camera.m_forward = glm::vec3(0.f, 0.f, -1.f);
	camera.m_worldUp = glm::vec3(0.f, 1.f, 0.f);
	camera.m_fov = 70.f;
	camera.m_screenHeight = float(windowWidth);
	camera.m_screenWidth = float(windowHeight);
	camera.m_nearClip = 0.1f;
	camera.m_farClip = 1000.f;
	camera.m_pitch = 0.f;
	camera.m_yaw = -90.f;
	camera.UpdateBasisAxis();

	camera.m_invViewProj = camera.GetInverseViewMatrix() * camera.GetInverseProjectionMatrix();

	GLFWViewer* viewer = new GLFWViewer(windowWidth, windowHeight, pixels);
	//viewer->Create();

	state.d_camera = nullptr;
	hipMalloc((void**)&(state.d_camera), sizeof(Camera));
	cudaCheckErrors("hipMalloc camera fail");

	int maxIterations = 1;

	while (!glfwWindowShouldClose(viewer->m_window))
	{
		processInput(viewer->m_window, camera, pixels);
		camera.m_invViewProj = camera.GetInverseViewMatrix() * camera.GetInverseProjectionMatrix();

		//
		// EXECUTE CUDA KERNEL ON RENDER BUFFER
		//

		hipGraphicsMapResources(1, &viewer->interop->cgr[viewer->interop->index], viewer->stream);
		{
			viewer->cuda_err = pxl_kernel_launcher(viewer->interop->ca[viewer->interop->index] ,
				windowWidth,
				windowHeight,
				viewer->event,
				viewer->stream,
				state.d_geometry, camera, geometries.size(), maxIterations);
		}
		hipGraphicsUnmapResources(1, &viewer->interop->cgr[viewer->interop->index], viewer->stream);

		std::string title = "Firefly";
		glfwSetWindowTitle(viewer->m_window, title.c_str());

		//
		// BLIT & SWAP FBO
		// 
		glBlitNamedFramebuffer(viewer->interop->fb[viewer->interop->index], 0,
			0, 0, viewer->interop->width, viewer->interop->height,
			0, viewer->interop->height, viewer->interop->width, 0,
			GL_COLOR_BUFFER_BIT,
			GL_NEAREST);

		const GLfloat clear_color[] = { 0.0f, 0.0f, 0.0f, 0.0f };
		glClearNamedFramebufferfv(viewer->interop->fb[viewer->interop->index], GL_COLOR, 0, clear_color);
		// pxl_interop_clear(interop);
		viewer->interop->index = (viewer->interop->index + 1) % viewer->interop->count;


		glfwSwapBuffers(viewer->m_window);
		glfwPollEvents();
	}

	glfwDestroyWindow(viewer->m_window);
	glfwTerminate();

	hipFree(state.d_geometry);
	delete[] pixels;
	delete viewer;
	//delete triangleMeshGeometry;
	//hipFree(hostTriangleData);
	return 0;
}
